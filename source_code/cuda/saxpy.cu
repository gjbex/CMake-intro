#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

void print_array(float *x, int N) {
    for (int i = 0; i < N; i++) {
        printf("%7.3f", x[i]);
    }
    printf("\n");
}

__global__ void saxpy(float *y,
        const float a, const float *x, const long int N) {
    long int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i < N) {
        y[i] += a*x[i];
    }
}

int main(int argc, char *argv[]) {
    hipError_t err = hipSuccess;
    const int block_size = 32;
    const float alpha = 1.3f;
    int array_size = block_size; 
    if (argc > 1) {
        array_size = atoi(argv[1]);
    }
    float *x, *y;
    if (!(x = (float *) malloc(array_size*sizeof(float)))) {
        fprintf(stderr, "error: can not allocate x\n");
        exit(1);
    }
    if (!(y = (float *) malloc(array_size*sizeof(float)))) {
        fprintf(stderr, "error: can not allocate y\n");
        exit(1);
    }
    for (int i = 0; i < array_size; i++) {
        x[i] = i*0.01f;
        y[i] = 0.5f;
    }
    if (array_size < 100) {
        print_array(y, array_size);
        printf("%7.3f\n", alpha);
        print_array(x, array_size);
    }
    float *x_dev, *y_dev;
    err = hipMalloc(&x_dev, array_size*sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "error: could not allocate device array x\n");
        exit(2);
    }
    err = hipMemcpy(x_dev, x, array_size*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "error: could not copy to device array x\n");
        exit(2);
    }
    hipMalloc(&y_dev, array_size*sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "error: could not allocate device array y\n");
        exit(2);
    }
    err = hipMemcpy(y_dev, y, array_size*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "error: could not copy to device array y\n");
        exit(2);
    }
    saxpy<<<(array_size + block_size)/block_size, block_size>>>(y_dev, alpha, x_dev, array_size);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "error: could not compute saxpy: %s\n", hipGetErrorName(err));
        exit(3);
    }
    err = hipMemcpy(y, y_dev, array_size*sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "error: could not copy from device array y\n");
        exit(2);
    }
    if (array_size < 100) {
        print_array(y, array_size);
    }
    float max_error = 0.0f;
    for (int i = 0; i < array_size; i++) {
        float diff = fabs(y[i] - (0.5f + alpha*i*0.01f));
        if (diff > max_error) {
            max_error = diff;
        }
    }
    printf("max. error = %.7f\n", max_error);
    hipFree(x_dev);
    hipFree(y_dev);
    free(x);
    free(y);
    return 0;
}

